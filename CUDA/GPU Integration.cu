#include "hip/hip_runtime.h"


#include <iostream>
#define _USE_MATH_DEFINES
#include <math.h>
#include <time.h>

/*definition of benchmark*/
constexpr int limit = 10000;//integrate from -10000 to 10000
constexpr double dx = 0.0001;//the length of each interval
constexpr int N = limit / dx;//number of interval

#define block 16 //block(s) per gird
#define thread 32 //thread(s) per block
#define stride gridDim.x * blockDim.x

using namespace std;

/*since sinc funtion is even, only the value of sinc with x from 0 to 10000 is calculated, the sum will be doubled to get the final result */
__global__ void fx(double* d_f) {
	int Idx = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = Idx;i < N;i += stride) {
		double x = (i + 0.5) * dx;
		d_f[i] = sin(M_PI * x) / (M_PI * x);
	}
}

int main() {
	hipDeviceSynchronize();//synchronise CPU with GPU
	clock_t start = clock();//definition of timer
	
	double* f = new double[N];
	double* d_f;
	hipMalloc(&d_f, N * sizeof(double));
	fx<<<block, thread>>>(d_f);
	hipMemcpy(f, d_f, N * sizeof(double), hipMemcpyDeviceToHost);
	double sum = 0;
	for (int i = 0;i < N;i++) {
		sum += f[i];
	}
	hipFree(d_f);
	delete[] f;

	hipDeviceSynchronize();//synchronise CPU with GPU
	clock_t end = clock();

	/*the sum is doubled to get the final result*/
	cout << "the result is:" << 2 * sum * dx << "\n";
	cout << "the runtime is:" << end - start << "ms" << "\n";
	system("pause");
}
